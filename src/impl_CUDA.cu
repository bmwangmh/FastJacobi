#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#define d(i, j) ((i) * N + (j))
#define f(i, j) ((i) * N / 2 + ((j) / 2))
__global__ void kernel_col1(int N, double *p1, double *p2) {
  int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int j = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + 2 - i % 2;
  if (i < N - 1 && j < N - 1) {
      p1[f(i, j)] = (p2[f(i - 1, j)] + p2[f(i + 1, j)] + p2[f(i, j - 1)] + p2[f(i, j + 1)]) / 4.0f;
  }
}
__global__ void kernel_col2(int N, double *p1, double *p2) {
  int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int j = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + i % 2 + 1;
  if (i < N - 1 && j < N - 1) {
      p2[f(i, j)] = (p1[f(i - 1, j)] + p1[f(i + 1, j)] + p1[f(i, j - 1)] + p1[f(i, j + 1)]) / 4.0f;
  }
}
extern "C" void impl_CUDA(int N, int step, double *p) {
  hipSetDevice(0);
  double *p1 = (double*)calloc((N / 2 + 2) * N, sizeof(double));
  double *p2 = (double*)calloc((N / 2 + 2) * N, sizeof(double));
  double *d_p1, *d_p2;
  hipMalloc(&d_p1, (N / 2 + 2) * N * sizeof(double));
  hipMalloc(&d_p2, (N / 2 + 2) * N * sizeof(double));
  for(int i = 0; i < N; i++){
    for(int j = 0; j < N; j++){
      if((i + j) % 2 == 0) p1[f(i, j)] = p[d(i, j)];
      else p2[f(i, j)] = p[d(i, j)];
    }
  }
  hipMemcpy(d_p1, p1, (N / 2 + 2) * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_p2, p2, (N / 2 + 2) * N * sizeof(double), hipMemcpyHostToDevice);
  dim3 block(16, 16);
  dim3 grid((N - 1) / 2 / block.x + 1, (N - 2) / block.y + 1);
  for(int k = 1; k <= step; k++){
    if(k % 2){
      kernel_col1<<<grid, block>>>(N, d_p1, d_p2);
    }else{
      kernel_col2<<<grid, block>>>(N, d_p1, d_p2);
    }
    hipDeviceSynchronize();
  }
  hipMemcpy(p1, d_p1, (N / 2 + 2) * N * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(p2, d_p2, (N / 2 + 2) * N * sizeof(double), hipMemcpyDeviceToHost);
  hipFree(d_p1);
  hipFree(d_p2);
  for(int i = 0; i < N; i++){
    for(int j = 0; j < N; j++){
      if((i + j) % 2 == 0) p[d(i, j)] = p1[f(i, j)];
      else p[d(i, j)] = p2[f(i, j)];
    }
  }
  free(p1);
  free(p2);
}